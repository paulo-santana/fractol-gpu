#include "hip/hip_runtime.h"
#include "fractol.h"
#include <stdio.h>

void world_to_screen(
		t_data *fractol,
		long double world_x,
		long double world_y,
		int *screen_x,
		int *screen_y
		)
{
	*screen_x = (world_x - fractol->offset_x) * fractol->scale;
	*screen_y = (world_y - fractol->offset_y) * fractol->scale;
}

__device__ static void screen_to_world(
		t_data *fractol,
		int screen_x,
		int screen_y,
		double *world_x,
		double *world_y
		)
{
	*world_x = (screen_x / fractol->scale) + fractol->offset_x;
	*world_y = (screen_y / fractol->scale) + fractol->offset_y;
}

__global__ void calculate_mandelbrot(int *pixels, t_data *fractol) {
    size_t x = threadIdx.x;
    size_t y = blockIdx.x;
    int i = 0;
    int max_iter = fractol->max_iter;
    double x2 = 0, y2 = 0;
    double real = 0, imag = 0;
    double c_real, c_imag;
    size_t position = x + y * fractol->img.width;

    screen_to_world(fractol, x, y, &c_real, &c_imag);
    while (x2 + y2 <= 4 && i < max_iter)
    {
        imag = (real + real) * imag + c_imag;
        real = x2 - y2 + c_real;
        x2 = real * real;
        y2 = imag * imag;
        i++;
    }
    if (i == max_iter)
        pixels[position] = 0;
    else
        pixels[position] = fractol->colors[i];
}

__global__ void calculate_julia(int *pixels, t_data *fractol) {
    double real = 0,
           imag = 0,
           x2 = 0,
           y2 = 0;
    size_t x = threadIdx.x;
    size_t y = blockIdx.x;
    int position = y * fractol->img.width + x;
    int i = 0;

    real = x / fractol->scale + fractol->offset_x;
    imag = y / fractol->scale + fractol->offset_y;
    x2 = real * real;
    y2 = imag * imag;
    while (x2 + y2 <= 4 && i < fractol->max_iter)
    {
        imag = (real + real) * imag + fractol->c_imag;
        real = x2 - y2 + fractol->c_real;
        x2 = real * real;
        y2 = imag * imag;
        i++;
    }
    if (i == fractol->max_iter)
        pixels[position] = 0;
    else
        pixels[position] = fractol->colors[i];
}

extern "C"
void mandelbrot(t_data *fractol) {
    int *d_data;
    t_data *d_fractol;

    int size = fractol->img.width * fractol->img.height;
    int data_size = size * sizeof(int);
    hipMalloc((void **)&d_data, data_size);
    hipMalloc((void **)&d_fractol, sizeof(t_data));

    hipMemcpy(d_fractol, fractol, sizeof(t_data), hipMemcpyHostToDevice);

    /* calculate_mandelbrot<<<fractol->img.height, fractol->img.width>>>(d_data, d_fractol); */
    calculate_mandelbrot<<<fractol->img.height, fractol->img.width>>>(d_data, d_fractol);

    hipMemcpy(fractol->img.data, d_data, data_size, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipFree(d_fractol);
    hipDeviceSynchronize();
}
